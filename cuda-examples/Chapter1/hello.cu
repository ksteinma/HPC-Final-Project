
#include <hip/hip_runtime.h>
#include "stdio.h"

__global__ void helloFromGPU(void) {
	printf("Hello World from thread %d\n", threadIdx.x);
}

int main()
{
    printf("Hello World from CPU\n");
    helloFromGPU<<<10, 10>>>();
    hipDeviceReset();
    // cudaDeviceSynchronize();
    return 0;
}
